
#include <hip/hip_runtime.h>
__global__ void sinewave_vbo_kernel(float4 *pos,unsigned int width,unsigned int height,float animTime)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    float u = x/(float)width;
    float v = y/(float)height;
    u = (u * 2.0) - 1.0;
    v = (v * 2.0) - 1.0;
    float frequency = 4.0;
    float w = sinf(frequency * u + animTime) * cosf(frequency * v + animTime) * 0.5;
    pos[y * width + x] = make_float4(u,w,v,1.0);

}


void launchCudaKernel(float4 *pos,unsigned int Mesh_Width,unsigned int Mesh_Height,float Time)
{
    dim3 block(8,8,1);
    dim3 grid(Mesh_Width/block.x,Mesh_Height/block.y);
    sinewave_vbo_kernel<<<grid,block>>>(pos,Mesh_Width,Mesh_Height,Time);

}