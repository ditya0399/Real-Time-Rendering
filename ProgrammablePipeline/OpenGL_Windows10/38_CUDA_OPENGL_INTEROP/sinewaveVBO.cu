
#include <hip/hip_runtime.h>
__global__ void sinewave_vbo_kernel(float4 *pos,unsigned int width,unsigned int height,float time, uchar4 *colorPos)
{
	unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	// calculate uv coordinates
	float u = x / (float)width;
	float v = y / (float)height;
	u = u * 2.0f - 1.0f;
	v = v * 2.0f - 1.0f;

	// calculate simple sine wave pattern
	float freq = 4.0f;
	float w = sinf(u*freq + time) * cosf(v*freq + time) * 0.5f;

	// write output vertex
	pos[y*width + x] = make_float4(u, w, v, 1.0f);

	colorPos[y*width + x].w = 0;
	colorPos[y*width + x].x = 255.f *0.5*(1.f + sinf(w + x));
	colorPos[y*width + x].y = 255.f *0.5*(1.f + sinf(x)*cosf(y));
	colorPos[y*width + x].z = 255.f *0.5*(1.f + sinf(w + time / 10.f));


}


void launchCudaKernel(float4 *pos,unsigned int Mesh_Width,unsigned int Mesh_Height,float Time, uchar4 *colorPos)
{
    dim3 block(8,8,1);
    dim3 grid(Mesh_Width/block.x,Mesh_Height/block.y,1);
    sinewave_vbo_kernel<<<grid,block>>>(pos,Mesh_Width,Mesh_Height,Time,colorPos);

}